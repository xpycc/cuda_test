
#include <hip/hip_runtime.h>
__device__ int N;

__global__ void set_N(int n) {
	N = n;
}

__global__ void dev_add(int *a, int *b, int *c) {
	int id = blockIdx.x;
	if (id < N)
		c[id] = a[id] + b[id];
}

void add(int a[], int b[], int c[], int n) {
	int *dev_a, *dev_b, *dev_c;
	set_N<<<1, 1>>>(n);
	hipMalloc(&dev_a, n * sizeof(int));
	hipMalloc(&dev_b, n * sizeof(int));
	hipMalloc(&dev_c, n * sizeof(int));
	hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
	dev_add<<<n, 1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

__global__ void dev_add_n(int *a, int *b, int *c, int N) {
	int id = blockIdx.x;
	if (id < N)
		c[id] = a[id] + b[id];
}

void add_n(int a[], int b[], int c[], int n) {
	int *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, n * sizeof(int));
	hipMalloc(&dev_b, n * sizeof(int));
	hipMalloc(&dev_c, n * sizeof(int));
	hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(int), hipMemcpyHostToDevice);
	dev_add_n<<<n, 1>>>(dev_a, dev_b, dev_c, n);
	hipMemcpy(c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
