
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void dev_add_n(float *a, float *b, float *c, int n) {
	__shared__ float tmp[N];
	int id = threadIdx.x;
	if (id < N / 2) {
		tmp[id] = a[id] + b[id];
		__syncthreads();
	}
	c[id] = tmp[id];
}

void add_n(float a[], float b[], float c[], int n) {
	float *dev_a, *dev_b, *dev_c;
	hipMalloc(&dev_a, n * sizeof(float));
	hipMalloc(&dev_b, n * sizeof(float));
	hipMalloc(&dev_c, n * sizeof(float));
	hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);
	dev_add_n<<<1, n>>>(dev_a, dev_b, dev_c, n);
	hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

int main() {
	float a[N] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9},
		b[N] = {0, 1, 2 ,3, 4, 5, 6 ,7, 8, 9},
		c[N] = {0};
	add_n(a, b, c, N);
	for (int i = 0; i < N; ++i)
		printf("c[%d] = %.3f%c", i, c[i], i + 1 == N ? '\n' : ' ');
}
