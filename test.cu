#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main() {
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	add<<<1, 1>>>(2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2 + 7 = %d\n", c);
	hipFree(dev_c);
	hipGetDeviceCount(&c);
	printf("cuda device count: %d\n", c);
	for (int i = 0; i < c; ++i) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  %s\n", prop.deviceOverlap ? "Enabled" : "Disabled");
		printf("Kernel execution timeout:  %s\n", prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled");
		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Threads in wrap:  %d\n", prop.warpSize);
	}
	return 0;
}
