#include <stdio.h>

int main() {
	int c;
	hipGetDeviceCount(&c);
	printf("cuda device count: %d\n", c);
	for (int i = 0; i < c; ++i) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  %s\n", prop.deviceOverlap ? "Enabled" : "Disabled");
		printf("Kernel execution timeout:  %s\n", prop.kernelExecTimeoutEnabled ? "Enabled" : "Disabled");
		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture alignment:  %ld\n", prop.textureAlignment);
		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in wrap:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n",
				prop.maxThreadsDim[0], prop.maxThreadsDim[1],
				prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n",
				prop.maxGridSize[0], prop.maxGridSize[1],
				prop.maxGridSize[2]);
		puts("");
	}
	return 0;
}
